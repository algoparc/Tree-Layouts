#include "hip/hip_runtime.h"
#include "../src/bst-involutions.cuh"
#include "../src/query-bst.cuh"

int main(int argc, char *argv[]) {
    if (argc != 3) {
        fprintf(stderr,"Usage: %s <integer d such that n = 2^d - 1> <number of CPU threads>\n", argv[0]);
        exit(1);
    }

    uint32_t d = atoi(argv[1]);
    uint64_t n = pow(2, d) - 1;
    #ifdef DEBUG
    printf("n = 2^%d - 1 = %lu\n", d, n);
    #endif

    int p = atoi(argv[2]);
    omp_set_num_threads(p);

    uint64_t *A = (uint64_t *)malloc(n * sizeof(uint64_t));
    uint64_t *dev_A;
    hipMalloc(&dev_A, n * sizeof(uint64_t));

    double time[ITERS];

    //Construction
    initSortedList<uint64_t>(A, n);
    hipMemcpy(dev_A, A, n * sizeof(uint64_t), hipMemcpyHostToDevice);
    timePermuteBST<uint64_t>(dev_A, n, d, p);
    hipMemcpy(A, dev_A, n * sizeof(uint64_t), hipMemcpyDeviceToHost);

    //Querying
    uint64_t q = 1000000;
    while (q <= 100000000) {
        for (uint32_t i = 0; i < ITERS; ++i) {
            time[i] = timeQueryBST<uint64_t>(A, dev_A, n, q);
        }
        printQueryTimings(n, q, time); 
        
        if (q == 1000000) {
            q = 10000000;
        }
        else {
            q += 10000000;
        }
    }

    free(A);
    hipFree(dev_A);

    return 0;
}