#include "hip/hip_runtime.h"
#include "../src/veb-involutions.cuh"

int main(int argc, char **argv) {
    if (argc != 3) {
        fprintf(stderr, "Usage: %s <integer d such that n = 2^d - 1> <number of CPU threads>\n", argv[0]);
        exit(1);
    }

    uint32_t d = atoi(argv[1]);
    uint64_t n = pow(2, d) - 1;
    #ifdef DEBUG
    printf("n = 2^%d - 1 = %lu\n", d, n);
    #endif

    uint32_t p = atoi(argv[2]);
    omp_set_num_threads(p);

    uint64_t *A = (uint64_t *)malloc(n * sizeof(uint64_t));
    uint64_t *dev_A;
    hipMalloc(&dev_A, n * sizeof(uint64_t));

    double time[ITERS];

    //Construction
    for (uint32_t i = 0; i < ITERS; ++i) {
        initSortedList<uint64_t>(A, n);
        hipMemcpy(dev_A, A, n * sizeof(uint64_t), hipMemcpyHostToDevice);
        time[i] = timePermutevEB<uint64_t>(dev_A, n);
        hipMemcpy(A, dev_A, n * sizeof(uint64_t), hipMemcpyDeviceToHost);
    }
    printTimings(n, time);

    free(A);
    hipFree(A);

    return 0;
}