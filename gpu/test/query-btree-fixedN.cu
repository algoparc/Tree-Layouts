#include "../src/btree-cycles.cuh"
#include "../src/query-btree.cuh"

int main(int argc, char *argv[]){
    if (argc != 4) {
        fprintf(stderr,"Usage: %s <number of elements> <b value> <number of CPU threads>\n", argv[0]);
        exit(1);
    }

    uint64_t n = atol(argv[1]);
    uint64_t b = atoi(argv[2]);
    #ifdef DEBUG
    printf("n = %lu; b = %lu\n", n, b);
    #endif

    uint64_t p = atoi(argv[3]);
    omp_set_num_threads(p);

    uint64_t *A = (uint64_t *)malloc(n * sizeof(uint64_t));
    uint64_t *dev_A;
    hipMalloc(&dev_A, n * sizeof(uint64_t));

    double time[ITERS];

    //Construction
    initSortedList<uint64_t>(A, n);
    hipMemcpy(dev_A, A, n * sizeof(uint64_t), hipMemcpyHostToDevice);
    timePermuteBtree<uint64_t>(dev_A, n, b);
    hipMemcpy(A, dev_A, n * sizeof(uint64_t), hipMemcpyDeviceToHost);

    //Querying
    uint64_t q = 1000000;
    while (q <= 100000000) {
        for (uint32_t i = 0; i < ITERS; ++i) {
            time[i] = timeQueryBtree<uint64_t>(A, dev_A, n, b, q);
        }
        printTimings(q, time);

        if (q == 1000000) {
            q = 10000000;
        }
        else {
            q += 10000000;
        }
    }

    free(A);
    hipFree(A);

    return 0;
}
