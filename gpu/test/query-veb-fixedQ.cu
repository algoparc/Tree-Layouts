#include "hip/hip_runtime.h"
#include "../src/veb-cycles.cuh"
#include "../src/query-veb.cuh"

int main(int argc, char **argv) {
    if (argc != 3) {
        fprintf(stderr, "Usage: %s <number of queries> <number of CPU threads>\n", argv[0]);
        exit(1);
    }

    uint64_t q = atol(argv[1]);
    if (q <= 0) {
        fprintf(stderr, "Number of queries must be a positive integer\n");
    }

    uint32_t p = atoi(argv[2]);
    omp_set_num_threads(p);

    double time[ITERS];

    for (uint32_t d = 22; d <= 32; ++d) {
        uint64_t n = pow(2, d) - 1;
        #ifdef DEBUG
        printf("n = 2^%d - 1 = %lu\n", d, n);
        #endif

        uint64_t *A = (uint64_t *)malloc(n * sizeof(uint64_t));
        uint64_t *dev_A;
        hipMalloc(&dev_A, n * sizeof(uint64_t));

        //Construction
        initSortedList<uint64_t>(A, n);
        hipMemcpy(dev_A, A, n * sizeof(uint64_t), hipMemcpyHostToDevice);
        timePermutevEB<uint64_t>(dev_A, n);
        hipMemcpy(A, dev_A, n * sizeof(uint64_t), hipMemcpyDeviceToHost);

        //Build table used in querying
        vEB_table *table = (vEB_table *)calloc(d, sizeof(vEB_table));
        buildTable(table, n, d, 0);
        vEB_table *dev_table;
        hipMalloc(&dev_table, d * sizeof(vEB_table));
        hipMemcpy(dev_table, table, d * sizeof(vEB_table), hipMemcpyHostToDevice);

        //Querying
        for (uint32_t i = 0; i < ITERS; ++i) {
            time[i] = timeQueryvEB<uint64_t>(A, dev_A, dev_table, n, d, q);
        }
        printQueryTimings(n, q, time); 

        free(A);
        free(table);
        hipFree(dev_A);
        hipFree(dev_table);
    }

    return 0;
}