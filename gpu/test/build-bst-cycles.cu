#include "hip/hip_runtime.h"
/*
 * Copyright 2018-2020 Kyle Berney
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at

 *    http://www.apache.org/licenses/LICENSE-2.0

 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "../src/btree-cycles.cuh"

int main(int argc, char *argv[]) {
    if (argc != 3) {
        fprintf(stderr, "Usage: %s <integer d such that n = 2^d - 1> <number of CPU threads>\n", argv[0]);
        exit(1);
    }

    int d = atoi(argv[1]);
    int n = pow(2, d) - 1;
    #ifdef DEBUG
    printf("n = 2^%d - 1 = %lu\n", d, n);
    #endif

    int p = atoi(argv[2]);
    omp_set_num_threads(p);

    int *A = (int *)malloc(n * sizeof(int));
    int *dev_A;
    hipMalloc(&dev_A, n * sizeof(int));

    double time[ITERS];

    //Construction
    for (int i = 0; i < ITERS; ++i) {
        initSortedList<int>(A, n);
        hipMemcpy(dev_A, A, n * sizeof(int), hipMemcpyHostToDevice);
        time[i] = timePermuteBtree<int>(dev_A, n, 1);
        hipMemcpy(A, dev_A, n * sizeof(int), hipMemcpyDeviceToHost);
    }
    printTimings(n, time);

    free(A);
    hipFree(dev_A);

    return 0;
}