#include "hip/hip_runtime.h"
#include "../src/btree-cycles.cuh"

int main(int argc, char *argv[]) {
    if (argc != 3) {
        fprintf(stderr, "Usage: %s <integer d such that n = 2^d - 1> <number of CPU threads>\n", argv[0]);
        exit(1);
    }

    uint32_t d = atoi(argv[1]);
    uint64_t n = pow(2, d) - 1;
    #ifdef DEBUG
    printf("n = 2^%d - 1 = %lu\n", d, n);
    #endif

    int p = atoi(argv[2]);
    omp_set_num_threads(p);

    uint64_t *A = (uint64_t *)malloc(n * sizeof(uint64_t));
    uint64_t *dev_A;
    hipMalloc(&dev_A, n * sizeof(uint64_t));

    double time[ITERS];

    //Construction
    for (uint32_t i = 0; i < ITERS; ++i) {
        initSortedList<uint64_t>(A, n);
        hipMemcpy(dev_A, A, n * sizeof(uint64_t), hipMemcpyHostToDevice);
        time[i] = timePermuteBtree<uint64_t>(dev_A, n, 1);
        hipMemcpy(A, dev_A, n * sizeof(uint64_t), hipMemcpyDeviceToHost);
    }
    printTimings(n, time);

    free(A);
    hipFree(dev_A);

    return 0;
}