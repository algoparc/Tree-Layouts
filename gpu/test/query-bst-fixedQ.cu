#include "hip/hip_runtime.h"
/*
 * Copyright 2018-2020 Kyle Berney
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at

 *    http://www.apache.org/licenses/LICENSE-2.0

 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "../src/bst-involutions.cuh"
#include "../src/query-bst.cuh"

int main(int argc, char *argv[]) {
    if (argc != 3) {
        fprintf(stderr, "Usage: %s <number of queries> <number of CPU threads>\n", argv[0]);
        exit(1);
    }

    uint64_t q = atol(argv[1]);
    if (q <= 0) {
        fprintf(stderr, "Number of queries must be a positive integer\n");
    }
    
    int p = atoi(argv[2]);
    omp_set_num_threads(p);

    double time[ITERS];

    for (uint32_t d = 22; d <= 30; ++d) {
        uint64_t n = pow(2, d) - 1;
        #ifdef DEBUG
        printf("n = 2^%d - 1 = %lu\n", d, n);
        #endif

        uint64_t *A = (uint64_t *)malloc(n * sizeof(uint64_t));
        uint64_t *dev_A;
        hipMalloc(&dev_A, n * sizeof(uint64_t));

        //Construction
        initSortedList<uint64_t>(A, n);
        hipMemcpy(dev_A, A, n * sizeof(uint64_t), hipMemcpyHostToDevice);
        timePermuteBST<uint64_t>(dev_A, n);
        hipMemcpy(A, dev_A, n * sizeof(uint64_t), hipMemcpyDeviceToHost);

        //Querying
        for (uint32_t i = 0; i < ITERS; ++i) {
            time[i] = timeQueryBST<uint64_t>(A, dev_A, n, q);
        }
        printQueryTimings(n, q, time); 

        free(A);
        hipFree(dev_A);
    }

    return 0;
}