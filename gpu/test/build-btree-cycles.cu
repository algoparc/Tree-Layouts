#include "../src/btree-cycles.cuh"

int main(int argc, char *argv[]){
    if (argc != 4) {
        fprintf(stderr, "Usage: %s <number of elements> <b value> <number of CPU threads>\n", argv[0]);
        exit(1);
    }

    uint64_t n = atol(argv[1]);
    uint64_t b = atoi(argv[2]);
    #ifdef DEBUG
    printf("n = %lu; b = %lu\n", n, b);
    #endif

    uint64_t p = atoi(argv[3]);
    omp_set_num_threads(p);

    uint64_t *A = (uint64_t *)malloc(n * sizeof(uint64_t));
    uint64_t *dev_A;
    hipMalloc(&dev_A, n * sizeof(uint64_t));

    double time[ITERS];

    //Construction
    for (uint32_t i = 0; i < ITERS; ++i) {
        initSortedList<uint64_t>(A, n);
        hipMemcpy(dev_A, A, n * sizeof(uint64_t), hipMemcpyHostToDevice);
        time[i] = timePermuteBtree<uint64_t>(dev_A, n, b);
        hipMemcpy(A, dev_A, n * sizeof(uint64_t), hipMemcpyDeviceToHost);
    }
    printTimings(n, time);

    free(A);
    hipFree(A);

    return 0;
}