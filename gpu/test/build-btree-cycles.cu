/*
 * Copyright 2018-2021 Kyle Berney
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at

 *    http://www.apache.org/licenses/LICENSE-2.0

 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "../src/btree-cycles.cuh"

int main(int argc, char *argv[]){
    if (argc != 4) {
        fprintf(stderr, "Usage: %s <number of elements> <b value> <number of CPU threads>\n", argv[0]);
        exit(1);
    }

    uint64_t n = atol(argv[1]);
    uint64_t b = atoi(argv[2]);
    #ifdef DEBUG
    printf("n = %lu; b = %lu\n", n, b);
    #endif

    uint64_t p = atoi(argv[3]);
    omp_set_num_threads(p);

    uint32_t *A = (uint32_t *)malloc(n * sizeof(uint32_t));
    uint32_t *dev_A;
    hipMalloc(&dev_A, n * sizeof(uint32_t));

    double time[ITERS];

    //Construction
    for (uint32_t i = 0; i < ITERS; ++i) {
        initSortedList<uint32_t>(A, n);
        hipMemcpy(dev_A, A, n * sizeof(uint32_t), hipMemcpyHostToDevice);
        time[i] = timePermuteBtree<uint32_t>(dev_A, n, b);
        hipMemcpy(A, dev_A, n * sizeof(uint32_t), hipMemcpyDeviceToHost);
    }
    printTimings(n, time);

    free(A);
    hipFree(A);

    return 0;
}
