#include "hip/hip_runtime.h"
/*
 * Copyright 2018-2021 Kyle Berney
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at

 *    http://www.apache.org/licenses/LICENSE-2.0

 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "../src/mveb-cycles.cuh"
#include "../src/query-mveb.cuh"

int main(int argc, char **argv) {
    if (argc != 3) {
        fprintf(stderr, "Usage: %s <number of queries> <number of CPU threads>\n", argv[0]);
        exit(1);
    }

    uint64_t q = atol(argv[1]);
    if (q <= 0) {
        fprintf(stderr, "Number of queries must be a positive integer\n");
    }

    uint32_t p = atoi(argv[2]);
    omp_set_num_threads(p);

    double time[ITERS];

    uint64_t n[13] = {
        4194303,
        8388607,
        10000000,
        16777215,
        33554431,
        67108863,
        100000000,
        134217727,
        268435455,
        536870911,
        1000000000,
        1073741823,
        2147483647
    };

    for (uint32_t j = 0; j < 13; ++j) {
        uint32_t *A = (uint32_t *)malloc(n[j] * sizeof(uint32_t));
        uint32_t *dev_A;
        hipMalloc(&dev_A, n[j] * sizeof(uint32_t));

        //Construction
        initSortedList<uint32_t>(A, n[j]);
        hipMemcpy(dev_A, A, n[j] * sizeof(uint32_t), hipMemcpyHostToDevice);
        timePermutevEB<uint32_t>(dev_A, n[j]);
        hipMemcpy(A, dev_A, n[j] * sizeof(uint32_t), hipMemcpyDeviceToHost);

        //Build table(s) used in querying
        uint32_t d = log2(n[j]) + 1;
        if (n[j] != pow(2, d) - 1) {     //non-perfect tree

            uint32_t temp_d, root_d, leaf_d, inc_d;
            uint64_t temp_n, r, l, num_full, inc_n;
            float log_leaf;
            temp_n = n[j];
            temp_d = d;

            uint32_t num_tables = 1;
            uint32_t table_size = d;
            while (temp_d >= 2) {
                root_d = (temp_d - 2)/2 + 1;        //floor((d - 2)/2) + 1
                leaf_d = temp_d - root_d;           //ceil((d - 2)/2.) + 1

                log_leaf = log2((float)leaf_d);
                if (log_leaf - ((int)log_leaf) != 0) {      //Not a perfectly balanced vEB, i.e., d is not a power of 2
                    leaf_d = pow(2, ceil(log_leaf));
                    root_d = temp_d - leaf_d;
                }

                r = pow(2, root_d) - 1;         //number of elements in the root subtree
                l = pow(2, leaf_d) - 1;         //number of elements in the full leaf subtrees

                num_full = (temp_n - r) / l;        //number of full leaf subtrees
                inc_n = temp_n - r - num_full*l;    //number of nodes in the incomplete leaf subtree

                if (inc_n == 0) break;

                inc_d = log2(inc_n) + 1;       //depth of the incomplete leaf subtree

                ++num_tables;
                table_size += inc_d;
                temp_n = inc_n;
                temp_d = inc_d;

                if (temp_n == pow(2, inc_d) - 1) break;
            }

            if (num_tables == 1) {
                vEB_table *table = (vEB_table *)calloc(d, sizeof(vEB_table));
                buildTable(table, n[j], d, 0);
                vEB_table *dev_table;
                hipMalloc(&dev_table, d * sizeof(vEB_table));
                hipMemcpy(dev_table, table, d * sizeof(vEB_table), hipMemcpyHostToDevice);

                //Querying
                for (uint32_t i = 0; i < ITERS; ++i) {
                    time[i] = timeQueryvEB<uint32_t>(A, dev_A, dev_table, n[j], d, q);
                }
                printQueryTimings(n[j], q, time); 

                free(table);
                hipFree(dev_table);
            }
            else {
                vEB_table *tables = (vEB_table *)calloc(table_size, sizeof(vEB_table));
                buildTable(tables, n[j], d, 0);

                uint64_t *idx = (uint64_t *)malloc(num_tables * sizeof(uint64_t));
                idx[num_tables - 1] = n[j];

                uint32_t *table_idx = (uint32_t *)malloc(num_tables * sizeof(uint32_t));
                table_idx[0] = 0;

                temp_n = n[j];
                temp_d = d;
                for (uint32_t i = 1; i < num_tables; ++i) {
                    table_idx[i] = table_idx[i-1] + temp_d;

                    root_d = (temp_d - 2)/2 + 1;        //floor((d - 2)/2) + 1
                    leaf_d = temp_d - root_d;           //ceil((d - 2)/2.) + 1

                    log_leaf = log2((float)leaf_d);
                    if (log_leaf - ((int)log_leaf) != 0) {      //Not a perfectly balanced vEB, i.e., d is not a power of 2
                        leaf_d = pow(2, ceil(log_leaf));
                        root_d = temp_d - leaf_d;
                    }

                    r = pow(2, root_d) - 1;         //number of elements in the root subtree
                    l = pow(2, leaf_d) - 1;         //number of elements in the full leaf subtrees

                    num_full = (temp_n - r) / l;        //number of full leaf subtrees
                    inc_n = temp_n - r - num_full*l;    //number of nodes in the incomplete leaf subtree
                    inc_d = log2(inc_n) + 1;            //depth of the incomplete leaf subtree

                    buildTable(&tables[table_idx[i]], inc_n, inc_d, 0);

                    idx[i-1] = n[j] - inc_n;

                    temp_n = inc_n;
                    temp_d = inc_d;
                }

                vEB_table *dev_tables;
                hipMalloc(&dev_tables, table_size * sizeof(vEB_table));
                hipMemcpy(dev_tables, tables, table_size * sizeof(vEB_table), hipMemcpyHostToDevice);

                uint64_t *dev_idx;
                hipMalloc(&dev_idx, num_tables * sizeof(uint64_t));
                hipMemcpy(dev_idx, idx, num_tables * sizeof(uint64_t), hipMemcpyHostToDevice);

                uint32_t *dev_table_idx;
                hipMalloc(&dev_table_idx, num_tables * sizeof(uint32_t));
                hipMemcpy(dev_table_idx, table_idx, num_tables * sizeof(uint32_t), hipMemcpyHostToDevice);

                //Querying
                for (uint32_t i = 0; i < ITERS; ++i) {
                    time[i] = timeQueryvEB_nonperfect<uint32_t>(A, dev_A, dev_tables, dev_idx, dev_table_idx, n[j], d, num_tables, q);
                }
                printQueryTimings(n[j], q, time);

                free(tables);
                free(idx);
                free(table_idx);
                hipFree(dev_tables);
                hipFree(dev_idx);
                hipFree(dev_table_idx);
            }
        }
        else {      //perfect tree
            vEB_table *table = (vEB_table *)calloc(d, sizeof(vEB_table));
            buildTable(table, n[j], d, 0);
            vEB_table *dev_table;
            hipMalloc(&dev_table, d * sizeof(vEB_table));
            hipMemcpy(dev_table, table, d * sizeof(vEB_table), hipMemcpyHostToDevice);

            //Querying
            for (uint32_t i = 0; i < ITERS; ++i) {
                time[i] = timeQueryvEB<uint32_t>(A, dev_A, dev_table, n[j], d, q);
            }
            printQueryTimings(n[j], q, time);

            free(table);
            hipFree(dev_table);
        }

        free(A);
        hipFree(dev_A);
    }

    return 0;
}