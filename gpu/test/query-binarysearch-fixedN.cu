/*
 * Copyright 2018-2021 Kyle Berney
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at

 *    http://www.apache.org/licenses/LICENSE-2.0

 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "../src/binary-search.cuh"

int main(int argc, char *argv[]) {
    if (argc != 3) {
        fprintf(stderr, "Usage: %s <number of elements> <number of CPU threads>\n", argv[0]);
        exit(1);
    }

    uint64_t n = atol(argv[1]);
    int p = atoi(argv[2]);
    omp_set_num_threads(p);

    uint32_t *A = (uint32_t *)malloc(n * sizeof(uint32_t));
    uint32_t *dev_A;
    hipMalloc(&dev_A, n * sizeof(uint32_t));

    double time[ITERS];

    initSortedList<uint32_t>(A, n);
    hipMemcpy(dev_A, A, n * sizeof(uint32_t), hipMemcpyHostToDevice);

    //Querying
    uint64_t q = 1000000;
    while (q <= 100000000) {
        for (uint32_t i = 0; i < ITERS; ++i) {
            time[i] = timeQuery<uint32_t>(A, dev_A, n, q);
        }
        printQueryTimings(n, q, time); 
        
        if (q == 1000000) {
            q = 10000000;
        }
        else {
            q += 10000000;
        }
    }

    free(A);
    hipFree(dev_A);

    return 0;
}