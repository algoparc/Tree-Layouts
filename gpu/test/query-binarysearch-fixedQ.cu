#include "hip/hip_runtime.h"
#include "../src/binary-search.cuh"

int main(int argc, char *argv[]) {
    if (argc != 3) {
        fprintf(stderr, "Usage: %s <number of queries> <number of CPU threads>\n", argv[0]);
        exit(1);
    }

    uint64_t q = atol(argv[1]);
    if (q <= 0) {
        fprintf(stderr, "Number of queries must be a positive integer\n");
    }
    
    int p = atoi(argv[2]);
    omp_set_num_threads(p);

    double time[ITERS];

    for (uint32_t d = 22; d <= 32; ++d) {
        uint64_t n = pow(2, d) - 1;
        #ifdef DEBUG
        printf("n = 2^%d - 1 = %lu\n", d, n);
        #endif

        uint64_t *A = (uint64_t *)malloc(n * sizeof(uint64_t));
        uint64_t *dev_A;
        hipMalloc(&dev_A, n * sizeof(uint64_t));

        initSortedList<uint64_t>(A, n);
        hipMemcpy(dev_A, A, n * sizeof(uint64_t), hipMemcpyHostToDevice);
        
        //Querying
        for (uint32_t i = 0; i < ITERS; ++i) {
            time[i] = timeQuery<uint64_t>(A, dev_A, n, q);
        }
        printQueryTimings(n, q, time); 

        free(A);
        hipFree(dev_A);
    }

    return 0;
}