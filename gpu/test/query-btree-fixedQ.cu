#include "hip/hip_runtime.h"
/*
 * Copyright 2018-2020 Kyle Berney
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at

 *    http://www.apache.org/licenses/LICENSE-2.0

 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "../src/btree-cycles.cuh"
#include "../src/query-btree.cuh"

int main(int argc, char *argv[]){
    if (argc != 4) {
        fprintf(stderr, "Usage: %s <number of queries> <b value> <number of CPU threads>\n", argv[0]);
        exit(1);
    }

    uint64_t q = atol(argv[1]);
    if (q <= 0) {
        fprintf(stderr, "Number of queries must be a positive integer\n");
    }
    uint64_t b = atoi(argv[2]);

    uint64_t p = atoi(argv[3]);
    omp_set_num_threads(p);

    double time[ITERS];

    for (uint32_t d = 22; d <= 30; ++d) {
        uint64_t n = pow(2, d) - 1;
        #ifdef DEBUG
        printf("n = 2^%d - 1 = %lu; b = %lu\n", d, n, b);
        #endif

        uint64_t *A = (uint64_t *)malloc(n * sizeof(uint64_t));
        uint64_t *dev_A;
        hipMalloc(&dev_A, n * sizeof(uint64_t));

        //Construction
        initSortedList<uint64_t>(A, n);
        hipMemcpy(dev_A, A, n * sizeof(uint64_t), hipMemcpyHostToDevice);
        timePermuteBtree<uint64_t>(dev_A, n, b);
        hipMemcpy(A, dev_A, n * sizeof(uint64_t), hipMemcpyDeviceToHost);

        //Querying
        for (uint32_t i = 0; i < ITERS; ++i) {
            time[i] = timeQueryBtree<uint64_t>(A, dev_A, n, b, q);
        }
        printQueryTimings(n, q, time); 

        free(A);
        hipFree(dev_A);
    }

    return 0;
}
