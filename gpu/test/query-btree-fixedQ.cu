#include "hip/hip_runtime.h"
/*
 * Copyright 2018-2020 Kyle Berney
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at

 *    http://www.apache.org/licenses/LICENSE-2.0

 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "../src/btree-cycles.cuh"
#include "../src/query-btree.cuh"

int main(int argc, char *argv[]){
    if (argc != 4) {
        fprintf(stderr, "Usage: %s <number of queries> <b value> <number of CPU threads>\n", argv[0]);
        exit(1);
    }

    int q = atoi(argv[1]);
    if (q <= 0) {
        fprintf(stderr, "Number of queries must be a positive integer\n");
    }
    int b = atoi(argv[2]);

    int p = atoi(argv[3]);
    omp_set_num_threads(p);

    double time[ITERS];

    for (int d = 22; d <= 31; ++d) {
        int n = pow(2, d) - 1;
        #ifdef DEBUG
        printf("n = 2^%d - 1 = %d; b = %d\n", d, n, b);
        #endif

        int *A = (int *)malloc(n * sizeof(int));
        int *dev_A;
        hipMalloc(&dev_A, n * sizeof(int));

        //Construction
        initSortedList<int>(A, n);
        hipMemcpy(dev_A, A, n * sizeof(int), hipMemcpyHostToDevice);
        timePermuteBtree<int>(dev_A, n, b);
        hipMemcpy(A, dev_A, n * sizeof(int), hipMemcpyDeviceToHost);

        //Querying
        for (int i = 0; i < ITERS; ++i) {
            time[i] = timeQueryBtree<int>(A, dev_A, n, b, q);
        }
        printQueryTimings(n, q, time); 

        free(A);
        hipFree(dev_A);
    }

    return 0;
}
