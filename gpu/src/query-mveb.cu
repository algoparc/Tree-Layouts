#include "hip/hip_runtime.h"
/*
 * Copyright 2018-2020 Kyle Berney
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at

 *    http://www.apache.org/licenses/LICENSE-2.0

 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "query-veb.cuh"

//Builds table of size d = log_2(n) + 1, which is used to query the vEB
//Described in "Cache Oblivious Search Trees via Binary Trees of Small Height" (Brodal, Fagerberg, and Jacob. SODA 2002)
//vEB_table: table of size d to store B, T, and D values
//n: number of nodes in the current tree
//d: total depth in the current tree, i.e., n = 2^d - 1
//root_depth: depth of the current tree in the full tree
void buildTable(vEB_table *table, uint64_t n, uint32_t d, uint32_t root_depth) {
    if (n == 1) return;

    uint32_t leaf_d = floor((d - 2) / 2.) + 1;

    float log_leaf = log2((float)leaf_d);

    if (log_leaf - ((int)log_leaf) != 0) {      //Not a perfectly balanced vEB
        leaf_d = pow(2, ceil(log_leaf));
    }

    uint32_t root_d = d - leaf_d;

    uint64_t root_n = pow(2, root_d) - 1;
    uint64_t leaf_n = pow(2, leaf_d) - 1;

    uint32_t i = d - leaf_d;
    table[i].L = leaf_n;         //size of the bottom/leaf trees
    table[i].R = root_n;         //size of the top/root tree
    table[i].D = root_depth;     //depth of the corresponding top/root tree

    buildTable(table, root_n, root_d, root_depth);                      //recurse on top/root tree
    buildTable(&table[i], leaf_n, leaf_d, i + root_depth);              //recurse on bottom/leaf tree
}